
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void fill_value_kernel(int* res)
{
    *res = *res * 100;
}

extern "C" void fill_value(int* res)
{
    std::cout << "Entered function" << std::endl;
    int* res_;
    hipMallocManaged(&res_, sizeof(int));
    hipMemcpy(res_,res,sizeof(int),hipMemcpyHostToDevice);
    fill_value_kernel<<<1, 1>>>(res_);
    hipMemcpy(res,res_,sizeof(int),hipMemcpyDeviceToHost);
    hipFree(res_);
    std::cout << "Done." << std::endl;
}
