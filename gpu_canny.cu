#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core.hpp>
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cmath>

#define CHECK_BOUNDS(y, x, h, w) \
    ((x) < 0 || (y) < 0 || (x) >= (w) || (y) >= (h))

namespace cuda {
    __device__ int im2col(
            const float* src,
            const int src_w,
            const int src_h,
            const int k,
            const int y,
            const int x,
            float* dst
    ) {
        if (y + k > src_h || x + k > src_w || y < 0 || x < 0) {
            return 1;
        }

        const float* src_ = src + y * src_w + x;
        for (int i = 0; i < k; ++i) {
            for (int j = 0; j < k; ++j) {
                *dst++ = src_[j];
            }
            src_ += src_w;
        }

        return 0;
    }

    __device__ float dot_product(const float* src, const float* kernel, const int N) {
        float sum = 0;
        for (int i = 0; i < N; ++i) {
            sum += src[i] * kernel[i];
        }
        return sum;
    }

    __global__ void gradient_gpu_kernel(
            const float* src_,
            const int src_h,
            const int src_w,
            const int dst_h,
            const int dst_w,
            const float sigma,
            const int k,
            float* dst_,
            const float* kernelx,
            const float* kernely,
            unsigned char* directions) {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        if (CHECK_BOUNDS(i, j, dst_h, dst_w)) {
            printf("grad thread exit\n");
            return;
        }

        const int N = k * k;
        float sumx = 0;
        float sumy = 0;
        for (int ky = 0; ky < k; ++ky) {
            for (int kx = 0; kx < k; ++kx) {
                sumx += src_[(i + ky) * src_w + j + kx] * kernelx[ky * k + kx];
                sumy += src_[(i + ky) * src_w + j + kx] * kernely[ky * k + kx];
            }
        }
        dst_[i * dst_w + j] = sqrtf(sumx * sumx + sumy * sumy);
        float angle = atan2f(sumy, sumx);
        if (sumx != 0 || sumy != 0) {
            double dists[9] = {
                    fabs(angle - M_PI),
                    fabs(angle - 3 * M_PI_4),
                    fabs(angle - M_PI_2),
                    fabs(angle - M_PI_4),
                    fabs(angle),
                    fabs(angle + M_PI),
                    fabs(angle + 3 * M_PI_4),
                    fabs(angle + M_PI_2),
                    fabs(angle + M_PI_4)
            };
            int min_index = 0;
            double min = FLT_MAX;
            for (int el = 0; el < 8; ++el) {
                if (dists[el] < min) {
                    min = dists[el];
                    min_index = el;
                }
            }
            switch (min_index) {
                case 1:
                case 8:
                    directions[i * dst_w + j] = 255;
                    break;
                case 2:
                case 7:
                    directions[i * dst_w + j] = 128;
                    break;
                case 3:
                case 6:
                    directions[i * dst_w + j] = 192;
                    break;
                default:
                    directions[i * dst_w + j] = 64;
            }
        }
    }

    __global__ void zero_pad(
            const float* src,
            const int src_h,
            const int src_w,
            float* dst,
            const int dst_h,
            const int dst_w
    ) {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        int xoffset = (dst_h - src_h) / 2;
        int yoffset = (dst_w - src_w) / 2;
        if (CHECK_BOUNDS(i, j, src_h, src_w) || CHECK_BOUNDS(i + yoffset, j + xoffset, dst_h, dst_w)) {
            printf("padding thread exit\n");
            return;
        }

        dst[(i + yoffset) * dst_w + j + xoffset] = src[i * src_w + j];
    }

    __global__ void nonmax(
            const unsigned char* directions,
            const float* grad,
            const int grad_h,
            const int grad_w,
            const int dst_h,
            const int dst_w,
            float* dst
    ) {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        if (CHECK_BOUNDS(i, j, dst_h, dst_w)) {
            printf("thread exit\n");
            return;
        }

        float m = 0;
        switch (directions[i * dst_w + j]) {
            case 128:
                m = fmax(grad[i * grad_w + j + 1], grad[(i + 2) * grad_w + j + 1]);
                break;
            case 64:
                m = fmax(grad[(i + 1) * grad_w + j], grad[(i + 1) * grad_w + j + 2]);
                break;
            case 255:
                m = fmax(grad[i * grad_w + j + 2], grad[(i + 2) * grad_w + j]);
                break;
            case 192:
                m = fmax(grad[(i + 2) * grad_w + j + 2], grad[i * grad_w + j]);
                break;
        }

        if (grad[(i + 1) * grad_w + j + 1] > m) {
            dst[i * dst_w + j] = grad[(i + 1) * grad_w + j + 1];
        } else {
            dst[i * dst_w + j] = 0;
        }
    }

    __global__ void hysteresis(
            float* dst,
            const int dst_h,
            const int dst_w,
            const float low_thr,
            const float high_thr
    ) {
        int i = blockIdx.y * blockDim.y + threadIdx.y;
        int j = blockIdx.x * blockDim.x + threadIdx.x;
        if (CHECK_BOUNDS(i, j, dst_h, dst_w)) {
            printf("hyst bounds check\n");
        }

        int dst_index = i * dst_w + j;
        int neighbor_idxs[8] = {
                (i - 1) * dst_w + j - 1,
                (i - 1) * dst_w + j,
                (i - 1) * dst_w + j + 1,
                i * dst_w + j - 1,
                i * dst_w + j + 1,
                (i + 1) * dst_w + j - 1,
                (i + 1) * dst_w + j,
                (i + 1) * dst_w + j + 1
        };

        if (dst[dst_index] < low_thr) {
            dst[dst_index] = 0;
        } else if (dst[dst_index] < high_thr) {
            dst[dst_index] = 128;
        } else {
            dst[dst_index] = 255;
        }
        __syncthreads();

        __shared__ int changed;
        do {
            changed = 0;
            __syncthreads();

            if (dst[dst_index] == 128) {
                if (i > 0) {
                    if (j > 0) {
                        if (dst[neighbor_idxs[0]] == 255) {
                            dst[dst_index] = 255;
                        }
                    }
                    if (dst[neighbor_idxs[1]] == 255) {
                        dst[dst_index] = 255;
                    }
                    if (j < dst_w - 1) {
                        if (dst[neighbor_idxs[2]] == 255) {
                            dst[dst_index] = 255;
                        }
                    }
                }

                if (j > 0) {
                    if (dst[neighbor_idxs[3]] == 255) {
                        dst[dst_index] = 255;
                    }
                }
                if (j < dst_w - 1) {
                    if (dst[neighbor_idxs[4]] == 255) {
                        dst[dst_index] = 255;
                    }
                }

                if (i < dst_h - 1) {
                    if (j > 0) {
                        if (dst[neighbor_idxs[5]] == 255) {
                            dst[dst_index] = 255;
                        }
                    }
                    if (dst[neighbor_idxs[6]] == 255) {
                        dst[dst_index] = 255;
                    }
                    if (j < dst_w - 1) {
                        if (dst[neighbor_idxs[7]] == 255) {
                            dst[dst_index] = 255;
                        }
                    }
                }

                if (dst[dst_index] == 255) {
                    changed = 1;
                }
            }
            __syncthreads();
        } while (changed);

        if (dst[dst_index] == 128) {
            dst[dst_index] = 0;
        }
    }
}

void parseCudaResult(std::string label, hipError_t res) {
    if (res) {
        std::cout << label << ": " << hipGetErrorString(res) << std::endl;
    }
}

void allocKernel(const int k, const float sigma, float** kernelx, float** kernely) {
    parseCudaResult("kernelx alloc", hipMalloc(kernelx, k * k * sizeof(float)));
    parseCudaResult("kernely alloc", hipMalloc(kernely, k * k * sizeof(float)));

    float* kernelx_ = (float*) malloc(k * k * sizeof(float));
    float* kernely_ = (float*) malloc(k * k * sizeof(float));
    float* kx_ = kernelx_;
    float* ky_ = kernely_;
    float norm = -1.0 / (2.0 * M_PI * sigma * sigma * sigma * sigma);
    float expnorm = 2.0 * sigma * sigma;
    for (int i = 0; i < k; ++i) {
        float y = k / 2 - i;
        for (int j = 0; j < k; ++j) {
            float x = k / 2 - j;
            float scary = norm * std::exp(-(x * x + y * y) / expnorm);
            *kx_++ = x * scary;
            *ky_++ = y * scary;
        }
    }

    parseCudaResult("kernelx memcpy", hipMemcpy(*kernelx, kernelx_, k * k * sizeof(float), hipMemcpyHostToDevice));
    parseCudaResult("kernely memcpy", hipMemcpy(*kernely, kernely_, k * k * sizeof(float), hipMemcpyHostToDevice));
    free(kernelx_);
    free(kernely_);
}

void freeKernel(float* kernelx, float* kernely) {
    parseCudaResult("kernelx free", hipFree(kernelx));
    parseCudaResult("kernely free", hipFree(kernely));
}

extern "C" int canny_gpu(cv::Mat& src, const float sigma, const float low_thr, const float high_thr, cv::Mat& dst) {
    hipEvent_t all_start, exec_start, all_stop, exec_stop;
    float all_ms, exec_ms;
    hipEventCreate(&all_start);
    hipEventCreate(&exec_start);
    hipEventCreate(&all_stop);
    hipEventCreate(&exec_stop);
    hipEventRecord(all_start);

    parseCudaResult("select device", hipSetDevice(0));

    cv::Mat src_ = src;
    if (!src.isContinuous() || (src.type() & CV_MAT_DEPTH_MASK) != CV_32F) {
        src.convertTo(src_, CV_32F);
    }
    cv::Mat _src_;
    const int k = (int) std::ceil(sigma) * 6 + 1;
    cv::copyMakeBorder(src_, _src_, k / 2, k / 2, k / 2, k / 2, CV_HAL_BORDER_REFLECT);

    const int src_h = _src_.size[0];
    const int src_w = _src_.size[1];
    if (k >= src_h + 1 || k >= src_w + 1) {
        return 1;
    }
    const int dst_h = src_h - k + 1;
    const int dst_w = src_w - k + 1;
    const int cuda_dst_h = dst_h + dst_h % BLOCK_SIZE;
    const int cuda_dst_w = dst_w + dst_w % BLOCK_SIZE;
    dst = cv::Mat(cuda_dst_h, cuda_dst_w, CV_32FC1);

    float* cuda_src;
    float* cuda_dst;
    unsigned char* cuda_directions;
    float* cuda_grads;
    float* kernelx;
    float* kernely;

    allocKernel(k, sigma, &kernelx, &kernely);

    parseCudaResult("malloc src", hipMalloc(&cuda_src, src_h * src_w * sizeof(float)));
    parseCudaResult("malloc dst", hipMalloc(&cuda_dst, cuda_dst_h * cuda_dst_w * sizeof(float)));
    parseCudaResult("malloc grads", hipMalloc(&cuda_grads, (cuda_dst_h + 2) * (cuda_dst_w + 2) * sizeof(float)));
    parseCudaResult("malloc directions", hipMalloc(&cuda_directions, cuda_dst_h * cuda_dst_w * sizeof(unsigned char)));
    parseCudaResult("memcpy src",
                    hipMemcpy(cuda_src, _src_.ptr(), src_h * src_w * sizeof(float), hipMemcpyHostToDevice));

    const dim3 grid_size(cuda_dst_w / BLOCK_SIZE, cuda_dst_h / BLOCK_SIZE);
    const dim3 block_size(BLOCK_SIZE, BLOCK_SIZE);
    hipDeviceSetLimit(hipLimitMallocHeapSize, 512 * 1024 * 1024);
    hipEventRecord(exec_start);
    cuda::gradient_gpu_kernel<<<grid_size, block_size>>>(
            cuda_src,
                    src_h, src_w,
                    cuda_dst_h, cuda_dst_w,
                    sigma, k,
                    cuda_dst,
                    kernelx,
                    kernely,
                    cuda_directions
    );
    cuda::zero_pad<<<grid_size, block_size>>>(
            cuda_dst,
                    cuda_dst_h,
                    cuda_dst_w,
                    cuda_grads,
                    cuda_dst_h + 2,
                    cuda_dst_w + 2
    );
    cuda::nonmax<<<grid_size, block_size>>>(
            cuda_directions,
                    cuda_grads,
                    cuda_dst_h + 2,
                    cuda_dst_w + 2,
                    cuda_dst_h,
                    cuda_dst_w,
                    cuda_dst
    );
    cuda::hysteresis<<<grid_size, block_size>>>(
            cuda_dst,
                    cuda_dst_h,
                    cuda_dst_w,
                    low_thr,
                    high_thr
    );
    parseCudaResult("record stop", hipEventRecord(exec_stop));

    freeKernel(kernelx, kernely);

    parseCudaResult("memcpy dst",
                    hipMemcpy(dst.ptr(), cuda_dst, cuda_dst_h * cuda_dst_w * sizeof(float), hipMemcpyDeviceToHost));
    dst = dst(cv::Rect(0, 0, dst_w, dst_h));

    parseCudaResult("free src", hipFree(cuda_src));
    parseCudaResult("free directions", hipFree(cuda_directions));
    parseCudaResult("free grads", hipFree(cuda_grads));
    parseCudaResult("free dst", hipFree(cuda_dst));

    parseCudaResult("record all stop", hipEventRecord(all_stop));
    parseCudaResult("device sync", hipEventSynchronize(all_stop));
    parseCudaResult("calc total elapsed time", hipEventElapsedTime(&all_ms, all_start, all_stop));
    parseCudaResult("calc execution time", hipEventElapsedTime(&exec_ms, exec_start, exec_stop));
    std::cout << "GPU timings" << std::endl << "    total time (ms): " << all_ms << std::endl
              << "    execution time (ms): " << exec_ms << std::endl << "    data copy time (ms): " << all_ms - exec_ms
              << std::endl;

    parseCudaResult("destroy event all_start", hipEventDestroy(all_start));
    parseCudaResult("destroy event exec_start", hipEventDestroy(exec_start));
    parseCudaResult("destroy event all_stop", hipEventDestroy(all_stop));
    parseCudaResult("destroy event exec_stop", hipEventDestroy(exec_stop));

    return 0;
}


